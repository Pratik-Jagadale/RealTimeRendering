// header files
// standard headers
#include <stdio.h>

// cuda headers
#include <hip/hip_runtime.h>

// global variables
const int iNumberOfArrayElements = 5;

float *hostInput1 = NULL;
float *hostInput2 = NULL;
float *hostOutput = NULL;

float *deviceInput1 = NULL;
float *deviceInput2 = NULL;
float *deviceOutput = NULL;

// cude kernel
__global__ void vecAddGPU(float *in1, float *in2, float *out, int len)
{
    // code
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len)
    {
        out[i] = in1[i] + in2[i];
    }
}

// entry point function
int main(void)
{
    // function prototype
    void cleanup(void);

    // variable declartions
    int size = iNumberOfArrayElements * sizeof(float);
    hipError_t result = hipSuccess;

    // code
    //  host memory allocation
    hostInput1 = (float *)malloc(size);
    if (hostInput1 == NULL)
    {
        printf("Host Memory Allocation is Failed for hostInput1 array. \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostInput2 = (float *)malloc(size);
    if (hostInput2 == NULL)
    {
        printf("Host Memory Allocation is Failed for hostInput2 array. \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostOutput = (float *)malloc(size);
    if (hostOutput == NULL)
    {
        printf("Host Memory Allocation is Failed for hostOutput array. \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // filling values into host arrays
    hostInput1[0] = 101.1;
    hostInput1[1] = 102.1;
    hostInput1[2] = 103.1;
    hostInput1[3] = 104.1;
    hostInput1[4] = 105.1;

    hostInput2[0] = 101.1;
    hostInput2[1] = 102.1;
    hostInput2[2] = 103.1;
    hostInput2[3] = 104.1;
    hostInput2[4] = 105.1;

    // device memory allocation
    result = hipMalloc((void **)&deviceInput1, size);
    if (result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceInput1 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceInput2, size);
    if (result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceInput1 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceOutput, size);
    if (result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceOutput array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // copy data from host arrays into device arrays
    result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Host to Device Data is failed for deviceInput1 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Host to Device Data is failed for deviceInput2 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid = dim3(iNumberOfArrayElements, 1, 1);
    dim3 dimBlock = dim3(1, 1, 1);

    // CUDA Kernel for vector Additio n
    vecAddGPU<<<dimGrid, dimBlock>>>(deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements);

    // copy data from device array intp host array
    result = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        printf("Device to host Dta copy is failed for hostOutput array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // vector addition
    for (int i = 0; i < iNumberOfArrayElements; i++)
    {
        printf("%f + %f = %f\n", hostInput1[i], hostInput2[i], hostOutput[i]);
    }

    // cleanupp
    cleanup();

    return (0);
}

void cleanup(void)
{
    // code
    if (deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = NULL;
    }

    if (deviceInput2)
    {
        hipFree(deviceInput2);
        deviceInput2 = NULL;
    }

    if (deviceInput1)
    {
        hipFree(deviceInput1);
        deviceInput1 = NULL;
    }

    if (hostOutput)
    {
        hipFree(hostOutput);
        hostOutput = NULL;
    }

    if (hostInput2)
    {
        hipFree(hostInput2);
        hostInput2 = NULL;
    }

    if (hostInput1)
    {
        hipFree(hostInput1);
        hostInput1 = NULL;
    }
}
