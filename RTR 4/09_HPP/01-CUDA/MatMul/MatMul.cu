#include "hip/hip_runtime.h"
// header files
// standard headers
#include <stdio.h>

// cuda headers
#include <hip/hip_runtime.h>
#include "helper_timer.h"
// macros
#define BLOCK_WIDTH 32

// global variables
int *hostA = NULL;
int *hostB = NULL;
int *hostC = NULL;
int *gold = NULL;

int *deviceA = NULL;
int *deviceB = NULL;
int *deviceC = NULL;

float timeOnGPU = 0.0f;
float timeOnCPU = 0.0f;

// cude kernel
__global__ void matMulGPU(int *A, int *B, int *C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
    // code
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    // code
    if ((row < numARows) && (column < numBColumns))
    {
        int value = 0.0f;
        for (int k = 0.0f; k < numAColumns; k++)
        {
            int a = A[row * numAColumns + k];
            int b = B[k * numBColumns + column];
            value += a * b;
        }
        C[row * numCColumns + column] = value;
    }
}

// entry point function
int main(void)
{
    // function prototype
    void InitA(int *, int, int);
    void InitB(int *, int, int);
    void matMulCPU(int *, int *, int *, int, int, int, int);
    void cleanup(void);

    // variable declartions
    int numARows = BLOCK_WIDTH;
    int numAColumns = BLOCK_WIDTH;
    int numBRows = BLOCK_WIDTH;
    int numBColumns = BLOCK_WIDTH;

    int numCRows = numARows;
    int numCColumns = numBColumns;

    int numGoldRows = numARows;
    int numGoldColumns = numBColumns;

    int sizeA = numARows * numAColumns * sizeof(int);
    int sizeB = numBRows * numBColumns * sizeof(int);
    int sizeC = numCRows * numCColumns * sizeof(int);

    int sizeGold = numGoldRows * numBColumns * sizeof(int);

    hipError_t result = hipSuccess;

    // code
    //  host memory allocation
    hostA = (int *)malloc(sizeA);
    if (hostA == NULL)
    {
        printf("Host Memory Allocation is Failed for hostA array. \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostB = (int *)malloc(sizeB);
    if (hostB == NULL)
    {
        printf("Host Memory Allocation is Failed for hostB array. \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostC = (int *)malloc(sizeC);
    if (hostC == NULL)
    {
        printf("Host Memory Allocation is Failed for hostC array. \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    gold = (int *)malloc(sizeGold);
    if (gold == NULL)
    {
        printf("Host Memory Allocation is Failed for gold array. \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // printing matrix dimension and sizes
    printf("The Dimension Of Matrix 'hostA' Are : %d x %d\n", numARows, numAColumns);
    printf("The Dimension Of Matrix 'hostB' Are : %d x %d\n", numBRows, numBColumns);
    printf("The Dimension Of Matrix 'hostC' Are : %d x %d\n", numCRows, numCColumns);

    printf("The Dimension Of Matrix 'gold' Are : %d x %d\n", numGoldRows, numGoldColumns);

    printf("Size Of Matrix hostA = %d\n", sizeA);
    printf("Size Of Matrix hostB = %d\n", sizeB);
    printf("Size Of Matrix hostC = %d\n", sizeC);

    printf("Size Of Matrix Gold = %d\n", sizeGold);

    // fill source matrices
    InitA(hostA, numARows, numAColumns);
    InitB(hostB, numBRows, numBColumns);

    // device memory allocation

    // device memory allocation
    result = hipMalloc((void **)&deviceA, sizeA);
    if (result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceB, sizeB);
    if (result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceB matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceC, sizeC);
    if (result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceC array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // copy data from host arrays into device arrays
    result = hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Host to Device Data is failed for deviceA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Host to Device Data is failed for deviceA array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // cuda kernel  configuration
    dim3 dimGrid = dim3(ceil((int)numBColumns / (int)BLOCK_WIDTH), ceil((int)numARows / (int)BLOCK_WIDTH), 1);
    dim3 dimBlock = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // CUDA Kernel for vector Multiplication
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    matMulGPU<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBColumns, numCColumns);

    sdkStopTimer(&timer);
    timeOnGPU = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    timer = NULL;

    // copy data from device array into host array
    result = hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        printf("Device to host Dta copy is failed for hostC array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // vector addition on host
    matMulCPU(hostA, hostB, gold, numARows, numAColumns, numBColumns, numCColumns);

    // comparison
    const float epsilon = 0.0000001f;

    int breakValue = -1;
    bool bAccuracy = true;
    for (int i = 0; i < numCRows * numCColumns; i++)
    {
        float val1 = gold[i];
        float val2 = hostC[i];

        if (fabs(val1 - val2) > epsilon)
        {
            bAccuracy = false;
            breakValue = i;
            break;
        }
    }

    char str[128];
    if (bAccuracy == false)
        sprintf(str, "Comparison of CPU and GPU Matrix Multiplication is not within accuracy of 0.000001 at array index %d", breakValue);
    else
        sprintf(str, "Comparison of CPU and GPU Matrix Multiplication is within accuracy of 0.000001");

    // Output
    printf("Time taken for Matrix  Multiplication On CPU = %.6f\n", timeOnCPU);
    printf("Time taken for Matrix  Multiplication On GPU = %.6f\n", timeOnGPU);
    printf("%s\n", str);

    // cleanupp
    cleanup();

    return (0);
}

void fillFloatArrayRandomNumber(float *arr, int len)
{
    // code
    const float fscale = 1.0f / (float)RAND_MAX;
    for (int i = 0; i < len; i++)
    {
        arr[i] = fscale * rand();
    }
}

void InitA(int *data, int row, int col)
{
    int num = 1;
    // code
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            *(data + i * col + j) = num;
            num++;
        }
    }
}

void InitB(int *data, int row, int col)
{
    int num = BLOCK_WIDTH;
    // code
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            *(data + i * col + j) = num;
            num--;
        }
    }
}

void matMulCPU(int *A, int *B, int *C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
    // code
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int i = 0; i < numARows; i++)
    {
        for (int j = 0; j < numBColumns; j++)
        {
            int value = 0.0f;
            for (int k = 0; k < numAColumns; k++)
            {
                int a = A[i * numAColumns + k];
                int b = B[k * numBColumns + j];
                value += a * b;
            }
            C[i * numCColumns + j] = value;
        }
    }

    sdkStopTimer(&timer);
    timeOnCPU = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    timer = NULL;
}

void cleanup(void)
{
    // code
    if (deviceC)
    {
        hipFree(deviceC);
        deviceC = NULL;
    }

    if (deviceB)
    {
        hipFree(deviceB);
        deviceB = NULL;
    }

    if (deviceA)
    {
        hipFree(deviceA);
        deviceA = NULL;
    }

    if (gold)
    {
        hipFree(gold);
        gold = NULL;
    }

    if (hostC)
    {
        free(hostC);
        hostC = NULL;
    }

    if (hostB)
    {
        free(hostB);
        hostB = NULL;
    }

    if (hostA)
    {
        free(hostA);
        hostA = NULL;
    }
}
