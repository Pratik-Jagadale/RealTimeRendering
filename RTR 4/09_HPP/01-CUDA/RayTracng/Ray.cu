#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define rnd(x) (x * rand() / RAND_MAX)
#define SPHERE 20
#define INF 2e30f

struct Sphere
{
    float r, g, b;
    float radius;
    float x, y, z;
    __device__ float hit(float ox, float oy, float *n)
    {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius)
        {
            float dz = sqrt(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrt(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

int main(void)
{
    // capture the start time
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    CPUBitmap bitmap(DIM, DIM);
}
