#include <stdio.h>

int main(void)
{
    //  fucntion declartions
    void PrintCUDADeviceProperties(void);

    // code
    PrintCUDADeviceProperties();
}

void PrintCUDADeviceProperties(void)
{
    // code
    printf("CUDA INFORMATION : \n");

    printf("-------------------------------------------------\n");

    hipError_t ret_cuda_rt;
    int dev_count;
    ret_cuda_rt = hipGetDeviceCount(&dev_count);
    if (ret_cuda_rt != hipSuccess)
    {
        printf("CUDA Runtime API Error - hipGetDeviceCount() Failed Due to %s.\n", hipGetErrorString(ret_cuda_rt));
    }
    else if (dev_count == 0)
    {
        printf("There Is no CUDA Supportred Device On This System.\n");
        return;
    }
    else
    {
        printf("Total Number of CUDA supporting GPU Device/Devices On This System : %d\n", dev_count);
        for (int i = 0; i < dev_count; i++)
        {
            hipDeviceProp_t dev_prop;
            int driverVersion = 0, runtimeVersion = 0;

            ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
            if (ret_cuda_rt != hipSuccess)
            {
                printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
                return;
            }
            printf("\n");
            hipDriverGetVersion(&driverVersion);
            hipRuntimeGetVersion(&runtimeVersion);
            printf("**** CUDA DRIVER AND RUNTIME INFORMATION ****\n");
            printf("-----------------------------------------------\n");
            printf("CUDA Driver Version                             : %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
            printf("CUDA Runtime Version                            : %d.%d\n", runtimeVersion / 1000, (runtimeVersion / 1000) / 10);
            printf("\n");

            printf("=======================================================\n");
            printf("**** GPU DEVICE GENERAL INFORMATION ***\n");
            printf("========================================================\n");
            printf("GPU Deivce Number                               : %d \n", i);
            printf("GPU Device Name                                 : %s \n", dev_prop.name);
            printf("GPU Device Compute Capability                   : %d.%d \n", dev_prop.major, dev_prop.minor);
            printf("GPU Device Clock Rate                           : %d \n", dev_prop.clockRate);
            printf("GPU Device Type                                 : ");
            if (dev_prop.integrated)
                printf("Integrated (On-Board)   \n");
            else
                printf("Discreate ( card ) \n");
            printf("\n");

            printf("========================================================\n");
            printf("**** GPU DEVICE MEMOMRY INFORMATION ***\n");
            printf("========================================================\n");
            printf("GPU Device Total Memory                         : %.0f GB = %.0f MB = %llu Bytes \n", ((float)dev_prop.totalGlobalMem / 1048576.0f) / 1024.0f, (float)dev_prop.totalGlobalMem / 1048576.0f, (unsigned long long)dev_prop.totalGlobalMem);
            printf("GPU Device Constant Memory                      :  %llu Bytes \n", (unsigned long long)dev_prop.totalConstMem);
            printf("GPU Device Shared memory Per SMProcessor        : %lu Bytes", (unsigned long)dev_prop.sharedMemPerBlock);
            printf("\n");

            printf("*** GPU DEVICE MULTIPROCESSOR INFORMATION ****\n");
            printf("=======================================================\n");
            printf("GPU Device Number Of SMProcessors               : %d \n", dev_prop.multiProcessorCount);
            printf("GPU Device number Of Registers Per SMProcessor  : %d \n", dev_prop.regsPerBlock);
            printf("\n");

            printf("*** GPU DEVICE THREAD INFORMATION ***\n");
            printf("======================================================\n");
            printf("GPU Device Maximum Number of Thread Per SMProcessor : %d\n", dev_prop.maxThreadsPerMultiProcessor);
            printf("GPU Device Maximum Number of Thread Per Block       : %d\n", dev_prop.maxThreadsPerBlock);
            printf("GPU Device Threads In Warp                          : %d\n", dev_prop.warpSize);
            printf("GPU Device Maximum Thread Dimensions                : %d , %d ,%d\n ", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);
            printf("\n");

            printf("*** GPU DEVICE DRIVER INFORMATION ***\n");
            printf("======================================================\n");
            printf("GPU Device has ECC Support                          : %s\n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
            printf("GPU DEVICE CUDA Driver Mode (TCC Or WDDM) %s \n", dev_prop.tccDriver ? "TCC (Tesala Compute Cluster Driver " : "WDDM (WINODWS Display Driver Model)");
#endif
            printf("**************************************************************************************\n");
        }
    }
}
